#include "hip/hip_runtime.h"
// this experimental version tries to allocate more in the cache size. This is to check how well it will behave for large transmitter arrays.
// since there is excellent locality in cache access, and the cache is prefetched, this may have a minimal impact

// version 5: try to resolve the first line problem
// the problem is that pixels (1,2) (1,3) and so on do not compute correctly. They appear to be shifted 1px to the right - why?
#include "mex.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define pi = 3.141592653589793f

// p2: try to use global constant memory

// -------------- the kernel runs on the GPU ---------------------------

//__device__ __constant__ float tx_const[1536];  // tx table limited to 256 entries - takes 6*256=1536 floats, which is 6144 bytes out of 8192 available
//  __device__ 
        __constant__ float tx_const[12288]; // allocate 64x64 transmitters
// i have noted that this line above actually depends on the driver version. The older driver requires NOT to put in the __device__ in, the newer does.
//hipMemcpyToSymbol(HIP_SYMBOL( "dc_ArraySize"), &nArr, sizeof(unsigned long) );        


__global__ void BeamsimLambertKernel ( float *tx, unsigned int tx_length, float *out,
        unsigned int n, float d, float r,
        float k)
{
    unsigned int offset=0;
    unsigned int ix,iy,itx=0;
    float pressure,distance,kd,pressure_re,pressure_im=0;
    float dist2=0;
    float dix,diy,diz,lambert_x,lambert_y,lambert_z=0;
	float xbase,ybase,rho2,rhoi,cosphi,sinphi,cosl,sinl=0;
    float xbase0=-sqrtf((float)2)+(float)1e-8;
    // calculate ix,iy from thread built-in variables
    ix = blockIdx.x * blockDim.x + threadIdx.x; 
    iy = blockIdx.y * blockDim.y + threadIdx.y;
    //ix=0; // debug // 
	//C// for (iy=0; iy<ny; iy++)
    //C//  for (ix=0; ix<nx; ix++)             
    
    // make sure that this thread won't try to calculate non-existent receiver
    if (iy>n) return;
    
    if (ix>n) return;
    
    // start actual calculation
        pressure_re=0;
        pressure_im=0;

		xbase=(float)ix*d+xbase0;
		ybase=(float)iy*d+xbase0; // it would be an optimisation not to recalculate it for each pixel, this has to stay here be due to future port to CUDA where each pixel has it's own thread		
        rho2=xbase*xbase+ybase*ybase;
        offset=ix+n*iy;
        if (rho2>(float)2)
        {
          out[offset]=0;
          return; 
        }
        rhoi=rsqrtf(rho2);
        cosl=-ybase*rhoi;
        cosphi=sqrtf(rho2-rho2*rho2/(float)4); 
        lambert_x=r*cosl*cosphi;
        sinl=xbase*rhoi;
        lambert_y=r*sinl*cosphi;
        sinphi=(float)1-rho2/(float)2;
        lambert_z=r*sinphi;
        for (itx=0; itx<tx_length*6; itx=itx+6) // this hopefully acesses the same memory location for each thread and therefore will be cached
         {
            //    distance=single(sqrt( (ix*dx+x0-tx(1,itx)).^2 + (iy*dy+y0-tx(2,itx)).^2 + (iz*dz+z0-tx(3,itx)).^2 ));
            dix=(lambert_x-tx_const[0+itx]);
            diy=(lambert_y-tx_const[1+itx]);
            diz=(lambert_z-tx_const[2+itx]);
            distance=sqrtf( dix*dix + diy*diy + diz*diz ); 
            // alternative version:
            //distance=hypotf(hypotf(dix,diy),diz); // turns out that this slows down the code nearly 2x while not providing better accuracy for all realistic parameter values.
            //mexPrintf("distance = %0.2f\n",distance);
            // DirectivityCos=single((iz*dz+z0-tx(3,itx)./distance));                            
            //C//OBSOLETE// directivitycos=diz/distance;
            // mexPrintf("DirCos = %0.3f\n",directivitycos);
            // kd=(-k*distance+tx(6,itx));
            kd=-k*distance+tx_const[5+itx];
            //mexPrintf("kd = %0.3f\n",kd);
            // pressure_re=pressure_re+cos(kd)*tx(5,itx)*DirectivityCos/(2*pi*distance);                
             //pressure_im=pressure_im+sin(kd)*tx(5,itx)*DirectivityCos/(2*pi*distance);
            
            // oryginal includes directivitycos
            // dist2=tx[4+itx]*directivitycos/(2*pi*distance);
            // *exclude directivitycos, assume ominidirectional until i settle the element directivity 
            dist2=tx_const[4+itx]/(6.283185307179586f*distance); //equals 2*pi
            
//             tmp=tx[4+itx];
//             mexPrintf("tx_amp=%0.2f\n",tmp);
            
 //           mexPrintf("dist2 = %0.3f\n",dist2);
            pressure_re=pressure_re+__cosf(kd)*dist2;                    
  //          mexPrintf("p_re=%0.3f\n",pressure_re);
            pressure_im=pressure_im+__sinf(kd)*dist2; 
            
            // note: __sinf is an simlpified sin function that yields less accurate result. May need to switch to full sin for final product, ok for testing for now
            
            // note 2: function sincosf(...) may be faster in this case - calculates both sin and cos. but since it requires additional accumulators, 
            // a detailed test will be required to find out what's faster.
            
          }        
        // mem write
        // out(ix+1,iy+1,iz+1)=abs(pressure_re+1i*pressure_im); 
        pressure=sqrtf(pressure_re*pressure_re+pressure_im*pressure_im);        
        // in CUDA, i need to calculate rx array memory offset manually for each thread:
	    //offset=ix+nx*iy+(ny*nx)*iz;        
        
       // mexPrintf("x = %d; y=%d; z=%d; offset=%d\n",ix,iy,iz,offset);
       out[offset]=(float)pressure; //left for debug
        //debug
       //out[offset]=(float)iy;
       //C// offset=offset++; //this shortcut only works for C                            
}


// ----------------- the MEX driver runs on the CPU --------------------
void mexFunction( int nlhs, mxArray *plhs[],
                  int nrhs, const mxArray *prhs[])
{
    
  //C//float *tx; // 5*n matrix: x,y,z,amplitude,phase
  //C//float *out; // m*
  float k,density,d,r,npts;
  unsigned int n;
  hipError_t error = hipSuccess;  
  int tmp_ct;
  //C//int totalsize;
  unsigned int tx_length;
  mwSize ndim;
  mwSize dims[2];
   /*  check for proper number of arguments */
  /* NOTE: You do not need an else statement when using mexErrMsgTxt
     within an if statement, because it will never get to the else
     statement if mexErrMsgTxt is executed. (mexErrMsgTxt breaks you out of
     the MEX-file) */
  if(nrhs!=4) 
    mexErrMsgTxt("4 inputs required.");
  if(nlhs!=1) 
    mexErrMsgTxt("One output required.");
  
  
  /* check to make sure the inputs 2-4 input argument are scalar singles */
  for (tmp_ct=1; tmp_ct<4; tmp_ct++)
  if( !mxIsSingle(prhs[tmp_ct]) || mxIsComplex(prhs[tmp_ct]) ||
      mxGetN(prhs[tmp_ct]) * mxGetM(prhs[tmp_ct])!=1 ) 
  {
    mexErrMsgTxt("Inputs 2-4 must be a scalar of class 'single'.");
  }
  
  /* check to make sure that input 1 is a n*6 single matrix */
  if (!mxIsSingle(prhs[0]) || mxIsComplex(prhs[0]) ||
          mxGetM(prhs[0])!=6)
  {
      mexErrMsgTxt("Input 1 must be 6*n matrix of class 'single'.");
  }
  
  // get variables from input
  tx_length=(unsigned int)mxGetN(prhs[0]);
  if (tx_length>2048)
      mexErrMsgTxt("Tx table is limited to 2048 entries for CUDA atm.");
  k=(float)mxGetScalar(prhs[1]);
  r=(float)mxGetScalar(prhs[2]);
  density=(float)mxGetScalar(prhs[3]);
  
  npts=ceilf(6.283185307179586*r/density);
  
  d=2*sqrtf((float)2)/npts; // distance between pixels in lambert map
  
  n=1+(unsigned int)ceilf(2*sqrtf(2)/d); // for some reason this fails to match with pure-C version if i don't add 1 here
 // totalsize = n*n;

  ndim=2;
  dims[0]=n; dims[1]=n;
  
  
  plhs[0] = mxCreateNumericArray(ndim,dims,mxSINGLE_CLASS,mxREAL);
          /* mwSize ndim, const mwSize *dims, 
         mxClassID classid, mxComplexity ComplexFlag */
  /*  create a C pointer to a copy of the output matrix */
  //C//out = (float *)mxGetPr(plhs[0]);
  //C//tx = (float *)mxGetPr(prhs[0]);
  
  //out[0]=(float)1; out[1]=(float)2; 
  // oryginal beamsim
  //beamsim(tx,tx_length,out,x0,y0,z0,nx,ny,nz,dx,dy,dz,k);
  
  // it is now time to allocate memory in gpu
  
  // first goes the tx array
  // this will go into constant memory later but resides in normal mem for now
  int tx_arraySize = tx_length * 6 ;
  int tx_memSize = sizeof(float) * tx_arraySize;
  float *d_tx;
    if ( hipMalloc( &d_tx, tx_memSize ) != hipSuccess )
        mexErrMsgTxt("Memory allocating failure on the GPU.Tx array");
  // plug in the tx array
  //OldMethod//
  if (
   hipMemcpy( d_tx, (float*) mxGetData(prhs[0]), tx_memSize, hipMemcpyHostToDevice)!=hipSuccess)
     mexErrMsgTxt("Tx Memory copy problem");
     
  //OldMethod - keep this for now, may not be needed later. Keep for now so that i don't need to change call pattern//
  
  error=hipMemcpyToSymbol(HIP_SYMBOL( "tx_const"), (float*) mxGetData(prhs[0]), tx_arraySize*sizeof(float) );
  if(error != hipSuccess){
		mexPrintf("Error is : %s\n", hipGetErrorString(hipGetLastError()));
		mexErrMsgTxt("Problem transferring tx_const table\n");
	}

  
  // now allocate rx array
  int rx_arraySize = n*n;
  int rx_memSize = sizeof(float) * rx_arraySize;
  float *d_rx;
  if ( hipMalloc( &d_rx, rx_memSize )  != hipSuccess )
        mexErrMsgTxt("Memory allocating failure on the GPU. Rx array");
  // run kernel
  // assume nz=1 and use 2D blocks, 2D grid
  
  dim3 threadsPerBlock(24,16); // threads in a block. 256 threads makes a typical occupancy. reduce if register pressure too high
  dim3 numBlocks((n+threadsPerBlock.x-1)/threadsPerBlock.x,(n+threadsPerBlock.y-1)/threadsPerBlock.y);  // blocks in a grid
  BeamsimLambertKernel<<< numBlocks, threadsPerBlock >>>( d_tx,tx_length,d_rx,n,d,r,k); 
   
  /* Get results back from the GPU and free device memory */
  hipMemcpy( (float*) mxGetData(plhs[0]), d_rx, rx_memSize, hipMemcpyDeviceToHost); 
  // finally
  hipFree( d_tx );
  hipFree( d_rx );
}